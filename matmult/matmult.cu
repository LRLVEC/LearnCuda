#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <random>
#include <_Time.h>
#include <_BLAS.h>

void generate_random_matrix(BLAS::mat& ref, std::mt19937 mt, float* data = nullptr, size_t row = 0, size_t col = 0)
{
	std::uniform_real_distribution<float> rd(-1, 1);
	BLAS::randomMat(ref, mt, rd);
	if (data)
	{
		for (size_t c0(0); c0 < row; ++c0)
		{
			for (size_t c1(0); c1 < col; ++c1)
			{
				data[row * c0 + c1] = ref(c0, c1);
			}
		}
	}
}

void write_to_float(float* mat_host, BLAS::mat& mat)
{
	for (size_t c0(0); c0 < mat.height; ++c0)
	{
		for (size_t c1(0); c1 < mat.width; ++c1)
		{
			mat_host[c0 * mat.width + c1] = mat(c0, c1);
		}
	}
}

void check(float* answer, BLAS::mat const& ref, size_t accu)
{
	BLAS::mat a(ref);
	for (size_t c0(0); c0 < a.height; ++c0)
	{
		for (size_t c1(0); c1 < a.width; ++c1)
		{
			a(c0, c1) = answer[a.width * c0 + c1];
		}
	}
	a -= ref;
	double eps(0);
	for (size_t c0(0); c0 < a.height; ++c0)
	{
		double rowEps(a.row(c0).norm1());
		eps += rowEps;
		// if (rowEps / (a.width * accu) > 1e-6)
		// {
		// 	printf("%d row eps: %e\n", c0, rowEps / (a.width * accu));
		// }
	}
	printf("Error: %e\n", eps / (a.width * a.height * accu));
}

constexpr unsigned int TILE_DIM = 32;

// calculates block(blockIdx.x, blockIdx.y) in the result matrix c
// note that a block's size is also TILE_DIM^2
__global__ void gemm(float* a, float* b, float* c, size_t a_x, size_t b_x)
{
	__shared__ float aTile[TILE_DIM][TILE_DIM], bTile[TILE_DIM][TILE_DIM];
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0.0f;
	for (int c0(0); c0 < a_x; c0 += TILE_DIM)
	{
		aTile[threadIdx.y][threadIdx.x] = a[row * a_x + c0 + threadIdx.x];
		bTile[threadIdx.y][threadIdx.x] = b[(c0 + threadIdx.y) * b_x + col];
		__syncthreads();
		for (int i = 0; i < TILE_DIM; i++)
		{
			// sum = __fmaf_rn(aTile[threadIdx.y][i], bTile[i][threadIdx.x], sum);

			//  ok!
			sum = __fmaf_ieee_rn(aTile[threadIdx.y][i], bTile[i][threadIdx.x], sum);
			// sum += aTile[threadIdx.y][i] * bTile[i][threadIdx.x];
		}
		__syncthreads();
	}
	c[row * b_x + col] = sum;
}


int main()
{
	std::mt19937 mt(114514);
	constexpr unsigned int a_row = 1024;
	constexpr unsigned int a_col = 1024;
	constexpr unsigned int b_row = 1024;
	constexpr unsigned int b_col = 1024;
	constexpr unsigned int c_row = a_row;
	constexpr unsigned int c_col = b_col;
	Timer timer;
	timer.begin();
	BLAS::mat a(a_col, a_row);
	BLAS::mat b(b_col, b_row);
	BLAS::mat c(c_col, c_row);
	timer.end();
	timer.print("malloc matrices:");

	timer.begin();
	generate_random_matrix(a, mt);
	timer.end();
	timer.print("gen rand mat a:");

	mt.discard(1llu << 20);

	timer.begin();
	generate_random_matrix(b, mt);
	timer.end();
	timer.print("gen rand mat b:");

	// timer.begin();
	// a(b, c);
	// timer.end();
	// timer.print("cpu mult");

	float* a_host;
	float* b_host;
	float* c_host;
	float* a_device;
	float* b_device;
	float* c_device;
	constexpr size_t a_size = sizeof(float) * a_row * a_col;
	constexpr size_t b_size = sizeof(float) * b_row * b_col;
	constexpr size_t c_size = sizeof(float) * c_row * c_col;

	timer.begin();
	a_host = (float*)malloc(a_size);
	b_host = (float*)malloc(b_size);
	c_host = (float*)malloc(c_size);
	timer.end();
	timer.print("malloc host:");

	timer.begin();
	hipMalloc(&a_device, a_size);
	hipMalloc(&b_device, b_size);
	hipMalloc(&c_device, c_size);
	timer.end();
	timer.print("malloc device:");

	write_to_float(a_host, a);
	write_to_float(b_host, b);

	timer.begin();
	hipMemcpy(a_device, a_host, a_size, hipMemcpyHostToDevice);
	hipMemcpy(b_device, b_host, b_size, hipMemcpyHostToDevice);
	hipMemset(c_device, 0, c_size);
	timer.end();
	timer.print("copy to device:");

	dim3 block = { TILE_DIM, TILE_DIM, 1 };
	dim3 grid = { c_col / TILE_DIM, c_row / TILE_DIM, 1 };
	printf("Launch grid: [%d, %d, %d]\n", grid.x, grid.y, grid.z);

	for (int c0(0); c0 < 10;++c0)
	{
		hipDeviceSynchronize();
		timer.begin();
		gemm << <grid, block >> > (a_device, b_device, c_device, a_col, b_col);
		hipDeviceSynchronize();
		timer.end();
		timer.print("cuda mult:");
	}

	// hipMemcpy(c_host, c_device, c_size, hipMemcpyDeviceToHost);
	// check(c_host, c, a_col);

	free(a_host);
	free(b_host);
	free(c_host);
	hipFree(a_device);
	hipFree(b_device);
	hipFree(c_device);

	// a.printToTableTxt("E:/files/C++/CUDA/LearnCuda/matmult/a.txt");
	// b.printToTableTxt("E:/files/C++/CUDA/LearnCuda/matmult/b.txt");
	// c.printToTableTxt("E:/files/C++/CUDA/LearnCuda/matmult/c.txt");

	return 0;
}
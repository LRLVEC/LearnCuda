#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <random>
#include <_Time.h>
#include <_BLAS.h>
#include <cutlass/gemm/device/gemm.h>

// All matrices are row-major

void generate_random_matrix(BLAS::mat& ref, std::mt19937 mt, float* data = nullptr, size_t row = 0, size_t col = 0)
{
	std::uniform_real_distribution<float> rd(-1, 1);
	BLAS::randomMat(ref, mt, rd);
	if (data)
	{
		for (size_t c0(0); c0 < row; ++c0)
		{
			for (size_t c1(0); c1 < col; ++c1)
			{
				data[row * c0 + c1] = ref(c0, c1);
			}
		}
	}
}

void write_to_float(float* mat_host, BLAS::mat& mat)
{
	for (size_t c0(0); c0 < mat.height; ++c0)
	{
		for (size_t c1(0); c1 < mat.width; ++c1)
		{
			mat_host[c0 * mat.width + c1] = mat(c0, c1);
		}
	}
}

void check(float* answer, BLAS::mat const& ref, size_t accu)
{
	BLAS::mat a(ref);
	for (size_t c0(0); c0 < a.height; ++c0)
	{
		for (size_t c1(0); c1 < a.width; ++c1)
		{
			a(c0, c1) = answer[a.width * c0 + c1];
		}
	}
	//a.print();
	//ref.print();
	a -= ref;
	//a.print();
	double eps(0);
	for (size_t c0(0); c0 < a.height; ++c0)
	{
		double rowEps(a.row(c0).norm1());
		eps += rowEps;
		// if (rowEps / (a.width * accu) > 1e-6)
		// {
		// 	printf("%d row eps: %e\n", c0, rowEps / (a.width * accu));
		// }
	}
	printf("Error: %e\n", eps / (a.width * a.height * accu));
}

constexpr unsigned int TILE_DIM = 16;

// calculates block(blockIdx.x, blockIdx.y) in the result matrix c
// note that a block's size is also TILE_DIM^2
__global__ void gemm(float* a, float* b, float* c, size_t a_x, size_t b_x)
{
	__shared__ float aTile[TILE_DIM][TILE_DIM], bTile[TILE_DIM][TILE_DIM];
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0.0f;
	for (int c0(0); c0 < a_x; c0 += TILE_DIM)
	{
		aTile[threadIdx.y][threadIdx.x] = a[row * a_x + c0 + threadIdx.x];
		bTile[threadIdx.y][threadIdx.x] = b[(c0 + threadIdx.y) * b_x + col];
		__syncthreads();
		for (int i = 0; i < TILE_DIM; i++)
		{
			// sum = __fmaf_rn(aTile[threadIdx.y][i], bTile[i][threadIdx.x], sum);

			//  ok!
			// aTile's read is broadcasted
			// bTile may suffer two way bank conflict when tile size is 16
			sum = __fmaf_ieee_rn(aTile[threadIdx.y][i], bTile[i][threadIdx.x], sum);
			// sum += aTile[threadIdx.y][i] * bTile[i][threadIdx.x];
		}
		__syncthreads();
	}
	c[row * b_x + col] = sum;
}

constexpr unsigned int VecSize_gemm_fast = 256;
constexpr unsigned int VecWarp_gemm_fast = 64;
constexpr unsigned int VecWarpNum_gemm_fast = VecSize_gemm_fast / VecWarp_gemm_fast;
constexpr unsigned int RollWidth_gemm_fast = 64;
constexpr unsigned int RollTimes_gemm_fast = VecWarp_gemm_fast;

__global__ void gemm_fast(float* a, float* b, float* c, size_t a_x, size_t b_x)
{
	// threadIdx.x: [0, RollWidth_gemm_fast - 1]
	// id: [0, VecSize_gemm_fast - 1]
	unsigned int id = threadIdx.x + threadIdx.y * VecWarp_gemm_fast;
	a += a_x * (blockIdx.x * VecSize_gemm_fast + id);
	b += b_x * threadIdx.y + blockIdx.y * RollWidth_gemm_fast + threadIdx.x;
	c += b_x * (blockIdx.x * VecSize_gemm_fast + id) + blockIdx.y * RollWidth_gemm_fast;
	__shared__ float bs[RollTimes_gemm_fast][RollWidth_gemm_fast + 1];
	float cs[RollWidth_gemm_fast] = { 0 };
	int cnt(0);
	do
	{
		// read: 8 + 64
		// calc: 4096
		// ratio: 4096 / (4*72) = 14.2
		for (int i(0); i < RollTimes_gemm_fast; i += VecWarpNum_gemm_fast)
		{
			bs[threadIdx.y + i][threadIdx.x] = b[i * b_x];
		}
		b += RollTimes_gemm_fast * b_x;
		cnt += RollTimes_gemm_fast;
		__syncthreads();
		for (int i(0); i < RollTimes_gemm_fast; ++i, ++a)
		{
			float a0 = a[0];
			for (int j(0); j < RollWidth_gemm_fast; ++j)
			{
				// slow! need to read one from shared each time:
				// fma.rn.f32     %f454, %f266, %f269, %f454;
				// ld.shared.f32  %f270, [%r23+16];
				// fma.rn.f32     %f453, %f266, %f270, %f453;
				// ld.shared.f32  %f271, [%r23+20];
				cs[j] += a0 * bs[i][j];
			}
		}
		__syncthreads();
	} while (cnt < a_x);
	for (int i(0); i < RollWidth_gemm_fast; ++i)
	{
		c[i] = cs[i];
	}
}


constexpr unsigned int TileSize = 128;
constexpr unsigned int RollLength = 16;
constexpr unsigned int KernelSize = 8;
constexpr unsigned int KernelLength = 8;
constexpr unsigned int KernelNum = TileSize / KernelSize;
constexpr unsigned int ThreadNum = KernelNum * KernelNum;

// launch: [32, 32, 1]
__global__ void gemm_faster(float* a, float* b, float* c, size_t a_x, size_t b_x)
{
	__shared__ float ta[TileSize][RollLength], tb[RollLength][TileSize];
	int row = blockIdx.y * blockDim.y * KernelSize;
	int col = blockIdx.x * blockDim.x * KernelSize;
	float ar[KernelSize][KernelLength];
	float br[KernelSize][KernelLength];
	float cr[KernelSize][KernelSize] = { 0 };
	for (int c0(0); c0 < a_x; c0 += RollLength)
	{
		// read: 128*16*2
		// calc: 128*128*16
		// ratio: 128/2 / 4 = 16
		int id = threadIdx.x + threadIdx.y * KernelNum;
		for (int c1(0); c1 < TileSize; c1 += ThreadNum / RollLength)
		{
			int x = id % RollLength;
			int y = c1 + id / RollLength;
			ta[y][x] = a[(row + y) * a_x + c0 + x];
		}
		for (int c1(0); c1 < RollLength; c1 += ThreadNum / TileSize)
		{
			int x = id % TileSize;
			int y = c1 + id / TileSize;
			tb[y][x] = b[(c0 + y) * b_x + col + x];
		}
		__syncthreads();
		for (int c1(0); c1 < RollLength; c1 += KernelLength)
		{
			for (int i(0); i < KernelSize; ++i)
			{
				for (int j(0); j < KernelLength; ++j)
				{
					ar[i][j] = ta[threadIdx.y * KernelSize + i][c1 + j];
					br[i][j] = tb[c1 + j][threadIdx.x * KernelSize + i];
				}
			}
			for (int i(0); i < KernelSize; ++i)
				for (int k(0); k < KernelLength; ++k)
					for (int j(0); j < KernelSize; ++j)
						cr[i][j] += ar[i][k] * br[j][k];
		}
		__syncthreads();
	}
	for (int c0(0); c0 < KernelSize; ++c0)
	{
		for (int c1(0); c1 < KernelSize; ++c1)
		{
			c[b_x * (row + threadIdx.y * KernelSize + c0) + col + threadIdx.x * KernelSize + c1] = cr[c0][c1];
		}
	}
}


hipError_t gemm_cutlass(float* a, float* b, float* c, size_t a_x, size_t b_x, size_t a_y)
{
	using RowMajor = cutlass::layout::RowMajor;
	using CutlassGemm = cutlass::gemm::device::Gemm<
		float, RowMajor,
		float, RowMajor,
		float, RowMajor>;

	CutlassGemm gemm_operator;
	CutlassGemm::Arguments args({ int(b_x), int(a_y), int(a_x) },
		{ a, a_x },
		{ b, b_x },
		{ c, b_x },
		{ c, b_x },
		{ 1.f, 0.f });
	cutlass::Status status = gemm_operator(args);
	if (status != cutlass::Status::kSuccess)
	{
		return hipErrorUnknown;
	}
	return hipSuccess;
}


int main()
{
	constexpr unsigned int loop_num(1);
	constexpr bool check_result(true);
	std::mt19937 mt(114514);
	constexpr unsigned int a_row = 2048;
	constexpr unsigned int a_col = 2048;
	constexpr unsigned int b_row = 2048;
	constexpr unsigned int b_col = 2048;
	constexpr unsigned int c_row = a_row;
	constexpr unsigned int c_col = b_col;
	printf("%dx%d * %dx%d -> %dx%d\n", a_row, a_col, b_row, b_col, c_row, c_col);
	Timer timer;
	timer.begin();
	BLAS::mat a(a_col, a_row);
	BLAS::mat b(b_col, b_row);
	BLAS::mat c(c_col, c_row);
	timer.end();
	timer.print("malloc matrices:");

	timer.begin();
	generate_random_matrix(a, mt);
	timer.end();
	timer.print("gen rand mat a:");

	mt.discard(1llu << 20);

	timer.begin();
	generate_random_matrix(b, mt);
	timer.end();
	timer.print("gen rand mat b:");

	if (check_result)
	{
		timer.begin();
		a(b, c);
		timer.end();
		timer.print("cpu mult");
	}

	float* a_host;
	float* b_host;
	float* c_host;
	float* a_device;
	float* b_device;
	float* c_device;
	constexpr size_t a_size = sizeof(float) * a_row * a_col;
	constexpr size_t b_size = sizeof(float) * b_row * b_col;
	constexpr size_t c_size = sizeof(float) * c_row * c_col;

	timer.begin();
	a_host = (float*)malloc(a_size);
	b_host = (float*)malloc(b_size);
	c_host = (float*)malloc(c_size);
	timer.end();
	timer.print("malloc host:");

	timer.begin();
	hipMalloc(&a_device, a_size);
	hipMalloc(&b_device, b_size);
	hipMalloc(&c_device, c_size);
	timer.end();
	timer.print("malloc device:");

	write_to_float(a_host, a);
	write_to_float(b_host, b);

	timer.begin();
	hipMemcpy(a_device, a_host, a_size, hipMemcpyHostToDevice);
	hipMemcpy(b_device, b_host, b_size, hipMemcpyHostToDevice);
	hipMemset(c_device, 0, c_size);
	timer.end();
	timer.print("copy to device:");

	dim3 block = { TILE_DIM, TILE_DIM, 1 };
	dim3 grid = { c_col / TILE_DIM, c_row / TILE_DIM, 1 };
	dim3 block_fast = { VecWarp_gemm_fast, VecWarpNum_gemm_fast, 1 };
	dim3 grid_fast = { c_row / VecSize_gemm_fast, c_col / RollWidth_gemm_fast, 1 };
	dim3 block_faster = { KernelNum, KernelNum, 1 };
	dim3 grid_faster = { c_row / TileSize, c_col / TileSize, 1 };
	printf("Launch grid: [%d, %d, %d]\n", grid.x, grid.y, grid.z);
	printf("Launch grid fast: [%d, %d, %d]\n", grid_fast.x, grid_fast.y, grid_fast.z);
	printf("Launch grid faster: [%d, %d, %d]\n", grid_faster.x, grid_faster.y, grid_faster.z);

	for (int c0(0); c0 < loop_num; ++c0)
	{
		hipDeviceSynchronize();
		timer.begin();
		gemm << <grid, block >> > (a_device, b_device, c_device, a_col, b_col);
		hipDeviceSynchronize();
		timer.end();
		timer.print("cuda mult:");
		printf("flops: %.3f T\n", double(a_col) * c_row * c_col / (timer.deltaT() * 1e12));
	}
	if (check_result)
	{
		hipMemcpy(c_host, c_device, c_size, hipMemcpyDeviceToHost);
		hipMemset(c_device, 0, c_size);
		check(c_host, c, a_col);
	}

	for (int c0(0); c0 < loop_num; ++c0)
	{
		hipDeviceSynchronize();
		timer.begin();
		gemm_fast << <grid_fast, block_fast >> > (a_device, b_device, c_device, a_col, b_col);
		hipDeviceSynchronize();
		timer.end();
		timer.print("cuda mult fast:");
		printf("flops: %.3f T\n", double(a_col) * c_row * c_col / (timer.deltaT() * 1e12));
	}
	if (check_result)
	{
		hipMemcpy(c_host, c_device, c_size, hipMemcpyDeviceToHost);
		hipMemset(c_device, 0, c_size);
		check(c_host, c, a_col);
	}

	for (int c0(0); c0 < loop_num; ++c0)
	{
		hipDeviceSynchronize();
		timer.begin();
		gemm_faster << <grid_faster, block_faster >> > (a_device, b_device, c_device, a_col, b_col);
		hipDeviceSynchronize();
		timer.end();
		timer.print("cuda mult faster:");
		printf("flops: %.3f T\n", double(a_col) * c_row * c_col / (timer.deltaT() * 1e12));
	}
	if (check_result)
	{
		hipMemcpy(c_host, c_device, c_size, hipMemcpyDeviceToHost);
		hipMemset(c_device, 0, c_size);
		check(c_host, c, a_col);
	}

	for (int c0(0); c0 < loop_num; ++c0)
	{
		hipDeviceSynchronize();
		timer.begin();
		gemm_cutlass(a_device, b_device, c_device, a_col, b_col, a_row);
		hipDeviceSynchronize();
		timer.end();
		timer.print("cuda mult cutlass:");
		printf("flops: %.3f T\n", double(a_col) * c_row * c_col / (timer.deltaT() * 1e12));
	}
	if (check_result)
	{
		hipMemcpy(c_host, c_device, c_size, hipMemcpyDeviceToHost);
		hipMemset(c_device, 0, c_size);
		check(c_host, c, a_col);
	}

	free(a_host);
	free(b_host);
	free(c_host);
	hipFree(a_device);
	hipFree(b_device);
	hipFree(c_device);

	// a.printToTableTxt("E:/files/C++/CUDA/LearnCuda/matmult/a.txt");
	// b.printToTableTxt("E:/files/C++/CUDA/LearnCuda/matmult/b.txt");
	// c.printToTableTxt("E:/files/C++/CUDA/LearnCuda/matmult/c.txt");

	return 0;
}